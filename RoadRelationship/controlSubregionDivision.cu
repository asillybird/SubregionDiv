#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// #include <cutil_inline.h>
#include <helper_functions.h>



//�꼰ȫ�ֱ�������
//////////////////////////////////////////////////////////////////////////
#define RowSize 5 //����5*5��lattice·��
#define NodeNumTol 25
// Do not consider the road beyond the lattice
#define RoadNum 40
#define NodeConnectNumMax 4

#define NumofRoads 4
// lanes for a road
#define NumofLanes 3

#define RoadSIZE 130 //��ʻ��·���������Ŀ
// in meter
#define VehicleLength 4

#define pi 3.1415926


#define NumofPhase 4  //ÿ��·����4���źŵƿ�����
// min for phase time
#define Cmin 120
// max for phase time
#define Cmax 240
// min green time
#define GreenTimeInPhaseMin 6
// max green time
#define GreenTimeInPhaseMax 90

#define flow 0.01

#define MaxGen 1000
#define PeopleNum 200

#define PCross 0.95
#define PMutate 0.05
/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/

struct VehicleAgent
{
	//�������
	// smaller number is in front
	int VehicleID;

	float position;//λ��
	float speed;//�ٶ�
	float acceleration;//���ٶ�
	bool VehicleFlag;//��ʾ����Ԫ�����Ƿ��г�
	float ExpSpeed;//�����ٶ�

	//�����ڵ���·����ʻ��ʱ
	int RoadTimeConsumed;
};

struct RoadAgent
{
	//��·���еĶ���
	VehicleAgent BufferRoad[RoadSIZE];//��������

	int RoadFront;//ͷָ��
	int RoadRear; //βָ��
	int RoadCount;//�����е�Ԫ�ظ���

};

struct TrafficLightsAgent
{
	//������̵�״̬
	// west and east go, then left, then south and north go, then left
	bool GreenLightStatus_Phase1;
	bool GreenLightStatus_Phase2;
	bool GreenLightStatus_Phase3;
	bool GreenLightStatus_Phase4;

	int cycle;//�źŵ�����

	int Phase;//���׼�źŵƵ���λ��

	//������̵Ƴ���ʱ��
	int GreenDuration__Phase1;
	int GreenDuration__Phase2;
	int GreenDuration__Phase3;
	int GreenDuration__Phase4;	
};

//��������·�����˵����ݽṹ
struct RoadNetworkResTemplate
{
	int NodeIndex;

	//���屾�ڵ��������·�������ڵ������Ľڵ����
	// 0, north, 1 west, 2 south, 3 east
	int NodeLinkTo[NumofRoads];
	// 0, south, 1 east, 2 north, 3 west
	int RoadLinkTo[NumofRoads];

	//���彻ͨ��
	TrafficLightsAgent TrafficLight;


	//�������ڼ���·�ι����ȵı���//////////////////////////////////
	//�������ݾ�Ϊ���복������
	float VehicleArrRate[NumofRoads];

	int RoadLength[NumofRoads];//ͨ�򱾽ڵ��·��·��	

	float RoadTraveTime[NumofRoads];//ͨ�򱾽ڵ��·�ϵĳ�������ʻʱ��
	
	int  VehicleInNumInAll[NumofRoads];//���򱾽ڵ�ĳ�������
	int RoadVehiclePassNum[NumofRoads][NumofLanes];//·��ͨ���ĳ�����Ŀ
	int  VehicleInNumInMax[NumofRoads];//���򱾽ڵ���������

	//�洢·������������
	float RoadRelationNum[NumofRoads];
	////////////////////////////////////////////////////////////////

	//������·�����ӵ�4����·
	RoadAgent Road[NumofRoads][NumofLanes];

};

RoadNetworkResTemplate RoadNetworkRes[NodeNumTol];
RoadNetworkResTemplate RoadNetworkResNew[NodeNumTol];


int SimulationClock;//�������ʱ��(s)
int SimulationDuration;//����������ʱ��(s)

//���ڵ������Ⱦ���
float LapMatirx[NodeNumTol][NodeNumTol];
float LapMatirx2[NodeNumTol][NodeNumTol];
float LapVector[RoadNum];

//��ͼ�õ����ݾ���
float Paint[NodeNumTol];

//�Ŵ��㷨�õ���Ⱦɫ�����
struct People
{
	int PeopleIndex;
	int Chomosome[RoadNum];
	int LapMatrixBinary[NodeNumTol][NodeNumTol];

	//������Ŀ
	int N;
	//�������������������--��໮��Ϊ25������ ��һ��·��һ������ ��ȫ����
	float NRelation[NodeNumTol][NodeNumTol];

	float fitness;
};
People PeopleCurrent[PeopleNum],PeopleNew[PeopleNum];


struct Tree
{
	int NodeIndex;
	int Leaf[4];
};

Tree tree[NodeNumTol];


bool NodeFlag[NodeNumTol];
bool LeafFlag[NodeNumTol];
int temp[NodeNumTol];
float temp2[NodeNumTol];
float temp3[NodeNumTol];

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif
/************************************************************************/
/* Example                                                              */
/************************************************************************/
__global__ static void HelloCUDA(char* result, int num)
{
	
}



void RoadNetworkInitial()
//����·������ ��ȷ���ӹ�ϵ
{	
	for (int i=0;i<NodeNumTol;i++)
	{
		
		RoadNetworkRes[i].NodeIndex=i;

		for (int k=0;k<NumofRoads;k++)
		{
			RoadNetworkRes[i].NodeLinkTo[k]=-1;	
			RoadNetworkRes[i].RoadLinkTo[k]=-1;	
		}		
	}

	for (int i=0;i<NodeNumTol;i++)
	{
		if (i%RowSize==0)
			//��Ӧ�����һ��Ԫ��
		{	
			if (i==0)
			{
				// starts from north 0, then west 1, then south 2, then east 3
				RoadNetworkRes[i].NodeLinkTo[0]=-1;
				// starts from south 0, then east 1, then north 2, then west 3
				RoadNetworkRes[i].RoadLinkTo[0]=-1;

				RoadNetworkRes[i].NodeLinkTo[1]=-1;
				RoadNetworkRes[i].RoadLinkTo[1]=-1;

				RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
				RoadNetworkRes[i].RoadLinkTo[2]=0;

				RoadNetworkRes[i].NodeLinkTo[3]=i+1;
				RoadNetworkRes[i].RoadLinkTo[3]=1;

			}
			else if (i==RowSize*(RowSize-1))
			{
				RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
				RoadNetworkRes[i].RoadLinkTo[0]=2;

				RoadNetworkRes[i].NodeLinkTo[1]=-1;
				RoadNetworkRes[i].RoadLinkTo[1]=-1;

				RoadNetworkRes[i].NodeLinkTo[2]=-1;
				RoadNetworkRes[i].RoadLinkTo[2]=-1;

				RoadNetworkRes[i].NodeLinkTo[3]=i+1;
				RoadNetworkRes[i].RoadLinkTo[3]=1;
			}
			else
			{
				RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
				RoadNetworkRes[i].RoadLinkTo[0]=2;

				RoadNetworkRes[i].NodeLinkTo[1]=-1;
				RoadNetworkRes[i].RoadLinkTo[1]=-1;

				RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
				RoadNetworkRes[i].RoadLinkTo[2]=0;

				RoadNetworkRes[i].NodeLinkTo[3]=i+1;
				RoadNetworkRes[i].RoadLinkTo[3]=1;

			}
		}
		else if (i%RowSize==RowSize-1)
			//��Ӧ�������һ��Ԫ��
		{

			if (i==RowSize-1)
			{
				RoadNetworkRes[i].NodeLinkTo[0]=-1;
				RoadNetworkRes[i].RoadLinkTo[0]=-1;

				RoadNetworkRes[i].NodeLinkTo[1]=i-1;
				RoadNetworkRes[i].RoadLinkTo[1]=3;

				RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
				RoadNetworkRes[i].RoadLinkTo[2]=0;

				RoadNetworkRes[i].NodeLinkTo[3]=-1;
				RoadNetworkRes[i].RoadLinkTo[3]=-1;	

			}
			else if (i==RowSize*RowSize-1)
			{
				RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
				RoadNetworkRes[i].RoadLinkTo[0]=2;

				RoadNetworkRes[i].NodeLinkTo[1]=i-1;
				RoadNetworkRes[i].RoadLinkTo[1]=3;

				RoadNetworkRes[i].NodeLinkTo[2]=-1;
				RoadNetworkRes[i].RoadLinkTo[2]=-1;

				RoadNetworkRes[i].NodeLinkTo[3]=-1;
				RoadNetworkRes[i].RoadLinkTo[3]=-1;	

			}
			else
			{
				RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
				RoadNetworkRes[i].RoadLinkTo[0]=2;

				RoadNetworkRes[i].NodeLinkTo[1]=i-1;
				RoadNetworkRes[i].RoadLinkTo[1]=3;

				RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
				RoadNetworkRes[i].RoadLinkTo[2]=0;

				RoadNetworkRes[i].NodeLinkTo[3]=-1;
				RoadNetworkRes[i].RoadLinkTo[3]=-1;	

			}
		}
		else if (i<RowSize && i>0)
			//��Ӧ�����һ��Ԫ��
		{
			RoadNetworkRes[i].NodeLinkTo[0]=-1;
			RoadNetworkRes[i].RoadLinkTo[0]=-1;

			RoadNetworkRes[i].NodeLinkTo[1]=i-1;
			RoadNetworkRes[i].RoadLinkTo[1]=3;

			RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
			RoadNetworkRes[i].RoadLinkTo[2]=0;

			RoadNetworkRes[i].NodeLinkTo[3]=i+1;
			RoadNetworkRes[i].RoadLinkTo[3]=1;	

		}
		else if (i<RowSize*RowSize-1 && i>RowSize*(RowSize-1))
			//��Ӧ�������һ��Ԫ��
		{
			RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
			RoadNetworkRes[i].RoadLinkTo[0]=2;

			RoadNetworkRes[i].NodeLinkTo[1]=i-1;
			RoadNetworkRes[i].RoadLinkTo[1]=3;

			RoadNetworkRes[i].NodeLinkTo[2]=-1;
			RoadNetworkRes[i].RoadLinkTo[2]=-1;

			RoadNetworkRes[i].NodeLinkTo[3]=i+1;
			RoadNetworkRes[i].RoadLinkTo[3]=1;	
		}
		else
			//�������Ե�������Ԫ��
		{
			RoadNetworkRes[i].NodeLinkTo[0]=i-RowSize;
			RoadNetworkRes[i].RoadLinkTo[0]=2;

			RoadNetworkRes[i].NodeLinkTo[1]=i-1;
			RoadNetworkRes[i].RoadLinkTo[1]=3;

			RoadNetworkRes[i].NodeLinkTo[2]=i+RowSize;
			RoadNetworkRes[i].RoadLinkTo[2]=0;

			RoadNetworkRes[i].NodeLinkTo[3]=i+1;
			RoadNetworkRes[i].RoadLinkTo[3]=1;	
		}
	}

	

}

void TrafficLightsDataInitial()
//��ͨ�����ݳ�ʼ��
{
	for (int l=0;l<NodeNumTol;l++)
	{
		//���ཻͨ�Ƴ�ʼ��
		RoadNetworkRes[l].TrafficLight.cycle=Cmax;

		RoadNetworkRes[l].TrafficLight.GreenDuration__Phase1=Cmax/4;
		RoadNetworkRes[l].TrafficLight.GreenDuration__Phase2=Cmax/4;
		RoadNetworkRes[l].TrafficLight.GreenDuration__Phase3=Cmax/4;
		RoadNetworkRes[l].TrafficLight.GreenDuration__Phase4=Cmax/4;

		RoadNetworkRes[l].TrafficLight.Phase=0;
		
		RoadNetworkRes[l].TrafficLight.GreenLightStatus_Phase1=false;
		RoadNetworkRes[l].TrafficLight.GreenLightStatus_Phase2=false;
		RoadNetworkRes[l].TrafficLight.GreenLightStatus_Phase3=false;
		RoadNetworkRes[l].TrafficLight.GreenLightStatus_Phase4=false;
	}

}

void RoadNetworkDataInitial()
//·�����ݳ�ʼ��
{
	for (int i=0;i<NodeNumTol;i++)
	{
		//if (i==12)
		{
			//Ϊ����·��������ֵ
			RoadNetworkRes[i].VehicleArrRate[0]=flow;
			
			RoadNetworkRes[i].VehicleArrRate[1]=flow;
		
			RoadNetworkRes[i].VehicleArrRate[2]=flow;
			
			RoadNetworkRes[i].VehicleArrRate[3]=flow;
			
		}
		//��һ��������--·�����ĵ�Ϊ������
		/*if(i==7)
		{
			RoadNetworkRes[i].VehicleArrRate[2]=flow;
		}
		if(i==11)
		{
			RoadNetworkRes[i].VehicleArrRate[3]=flow;
		}
		if(i==17)
		{
			RoadNetworkRes[i].VehicleArrRate[0]=flow;
		}
		if(i==13)
		{
			RoadNetworkRes[i].VehicleArrRate[1]=flow;
		}*/
		//�ڶ���������--·������Ϊ������
		
		
		//����·���ȸ�ֵ
		RoadNetworkRes[i].RoadLength[0]=RoadSIZE*VehicleLength;
		RoadNetworkRes[i].RoadLength[1]=RoadSIZE*VehicleLength;
		RoadNetworkRes[i].RoadLength[2]=RoadSIZE*VehicleLength;
		RoadNetworkRes[i].RoadLength[3]=RoadSIZE*VehicleLength;
		
		//����·ͨ���ĳ�����
		RoadNetworkRes[i].RoadVehiclePassNum[0][0]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[0][1]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[0][2]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[1][0]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[1][1]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[1][2]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[2][0]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[2][1]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[2][2]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[3][0]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[3][1]=0;
		RoadNetworkRes[i].RoadVehiclePassNum[3][2]=0;

		//����·����������ʻʱ��RoadTraveTime
		RoadNetworkRes[i].RoadTraveTime[0]=0;
		RoadNetworkRes[i].RoadTraveTime[1]=0;
		RoadNetworkRes[i].RoadTraveTime[2]=0;
		RoadNetworkRes[i].RoadTraveTime[3]=0;

		//����·���복����ֵ
		RoadNetworkRes[i].VehicleInNumInAll[0]=0;
		RoadNetworkRes[i].VehicleInNumInAll[1]=0;
		RoadNetworkRes[i].VehicleInNumInAll[2]=0;
		RoadNetworkRes[i].VehicleInNumInAll[3]=0;

		//����·�����������ֵ
		RoadNetworkRes[i].VehicleInNumInMax[0]=0;
		RoadNetworkRes[i].VehicleInNumInMax[1]=0;
		RoadNetworkRes[i].VehicleInNumInMax[2]=0;
		RoadNetworkRes[i].VehicleInNumInMax[3]=0;

		//����·�Ĺ���������
		RoadNetworkRes[i].RoadRelationNum[0]=0;
		RoadNetworkRes[i].RoadRelationNum[1]=0;
		RoadNetworkRes[i].RoadRelationNum[2]=0;
		RoadNetworkRes[i].RoadRelationNum[3]=0;
	}


	//��·�������ݸ�ֵ
	for (int i=0;i<NodeNumTol;i++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			for (int l=0;l<NumofLanes;l++)
			{
				RoadNetworkRes[i].Road[r][l].RoadFront=0;
				RoadNetworkRes[i].Road[r][l].RoadRear=0;
				RoadNetworkRes[i].Road[r][l].RoadCount=0;

				for (int v=0;v<RoadSIZE;v++)
				{
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].VehicleID=v;
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].RoadTimeConsumed=0;

					RoadNetworkRes[i].Road[r][l].BufferRoad[v].position=-1;
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].speed=0;
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].acceleration=0;
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].VehicleFlag=false;
					RoadNetworkRes[i].Road[r][l].BufferRoad[v].ExpSpeed=60/3.6+((80-30)/3.6)*((float)rand()/RAND_MAX);
				}
			}
		}
		
	}


	for (int i=0;i<NodeNumTol;i++)
	{
		for (int j=0;j<NodeNumTol;j++)
		{
			LapMatirx[i][j]=-1;
		}
	}

	for (int i=0;i<NodeNumTol;i++)
	{
		
		Paint[i]=0;
	
	}


}

void TrafficControlProGen2()
{
	for (int i=0;i<NodeNumTol;i++)
	{
		RoadNetworkRes[i].TrafficLight.cycle=Cmax;
		if (i==0)
		{
			RoadNetworkRes[i].TrafficLight.Phase=0;
		}
		else
		{
			RoadNetworkRes[i].TrafficLight.Phase=Cmax*((float)rand()/RAND_MAX);
		}
		
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1=Cmax/4;
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2=Cmax/4;
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3=Cmax/4;
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase4=Cmax/4;

	}
}

void TrafficControlProGen()  //���ɽ�ͨ�ƿ��Ʒ���
{
	srand(time(0));
	int C=Cmin+((Cmax-Cmin)*((float)rand()/RAND_MAX));
	for (int i=0;i<NodeNumTol;i++)
	{
		//float a=(float)rand()/RAND_MAX;
		
		RoadNetworkRes[i].TrafficLight.cycle=C;

		if (i==0)
		{
			RoadNetworkRes[i].TrafficLight.Phase=0;
		}
		else
		{
			RoadNetworkRes[i].TrafficLight.Phase=C*((float)rand()/RAND_MAX);
		}

		

		//��һ���̵Ƴ���ʱ��
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1=GreenTimeInPhaseMin+((GreenTimeInPhaseMax-GreenTimeInPhaseMin)*((float)rand()/RAND_MAX));
		
		//�ڶ����̵Ƴ���ʱ��
		int temp = RoadNetworkRes[i].TrafficLight.cycle - RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1;
		if (temp > GreenTimeInPhaseMax)
		{
			RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2 = GreenTimeInPhaseMin + ((GreenTimeInPhaseMax-GreenTimeInPhaseMin) * ((float)rand()/RAND_MAX));
		}
		else
		{
			RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2 = GreenTimeInPhaseMin + ((temp - GreenTimeInPhaseMin)*((float)rand()/RAND_MAX));
		}
		
		//�������̵Ƴ���ʱ��
		temp = RoadNetworkRes[i].TrafficLight.cycle-RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1-RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2;
		if (temp>GreenTimeInPhaseMax)
		{
			RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3=GreenTimeInPhaseMin+((GreenTimeInPhaseMax-GreenTimeInPhaseMin)*((float)rand()/RAND_MAX));
		}
		else
		{
			RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3=GreenTimeInPhaseMin+((temp-GreenTimeInPhaseMin)*((float)rand()/RAND_MAX));
		}

		//�������̵Ƴ���ʱ��
		int SumThree=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1+RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2+RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3;
		RoadNetworkRes[i].TrafficLight.GreenDuration__Phase4=RoadNetworkRes[i].TrafficLight.cycle - SumThree;
	}
}

void TrafficLightsRefresh()
//��ͨ���ź�ˢ��
{
	for (int i = 0; i < NodeNumTol; i++)
	{
		if(i==0)
		{
			int C=RoadNetworkRes[i].TrafficLight.cycle;
			int t1,t2,t3,t4;
			t1=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1;
			t2=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2;
			t3=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3;
			t4=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase4;

			if (SimulationClock%C>=0 && SimulationClock%C<=t1)
			{
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=true;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
			}
			if (SimulationClock%C>t1 && SimulationClock%C<=t1+t2)
			{
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=true;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
			}
			if (SimulationClock%C>t1+t2 && SimulationClock%C<=t1+t2+t3)
			{
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=true;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
			}
			if (SimulationClock%C>t1+t2+t3 && SimulationClock%C<=t1+t2+t3+t4)
			{
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=true;
			}
		}
		else
		{
			int C=RoadNetworkRes[i].TrafficLight.cycle;
			int phase=RoadNetworkRes[i].TrafficLight.Phase;
			int t1,t2,t3,t4;
			t1=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase1;
			t2=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase2;
			t3=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase3;
			t4=RoadNetworkRes[i].TrafficLight.GreenDuration__Phase4;

			if (SimulationClock<phase)
			{
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
				RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=true;
			}
			else
			{
				if ((SimulationClock-phase)%C>=0 && (SimulationClock-phase)%C<=t1)
				{
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=true;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
				}
				if ((SimulationClock-phase)%C>t1 && (SimulationClock-phase)%C<=t1+t2)
				{
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=true;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
				}
				if ((SimulationClock-phase)%C>t1+t2 && (SimulationClock-phase)%C<=t1+t2+t3)
				{
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=true;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=false;
				}
				if ((SimulationClock-phase)%C>t1+t2+t3 && (SimulationClock-phase)%C<=t1+t2+t3+t4)
				{
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase1=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase2=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase3=false;
					RoadNetworkRes[i].TrafficLight.GreenLightStatus_Phase4=true;
				}
			}
		}
	}
}

int VehGenEvent[NodeNumTol][NumofRoads][NumofLanes];//�������������¼���ʱ���
double interval[NodeNumTol][NumofRoads][NumofLanes];
int intervalReal[NodeNumTol][NumofRoads][NumofLanes];

float RandomGen(float Lambda)
{
	float Z;
	float output;

	do 
	{
		Z=((float)rand()/RAND_MAX);//ZΪ0-1֮����ȷֲ������
	} 
	while ((Z==0)||(Z==1));

	float Z1=log(Z);
	float X=1/Lambda;
	output=-X*Z1;

	return (output);
}

double GaussRandomGen(float mu, float sigma)
{
	float u1,u2;
	float T1,T2;
	float output1;
	float R=1000;
	while(R>=1.0||R==0)
	{
		do 
		{
			u1=((float)rand()/RAND_MAX);//u1Ϊ0-1֮����ȷֲ������
			u2=((float)rand()/RAND_MAX);//u2Ϊ0-1֮����ȷֲ������
		} 
		while (u1==u2);
		T1=2*u1-1;
		T2=2*u2-1;

		R=T1*T1+T2*T2;
	}
	float factor=sqrt((-2)*log(R)/R);
	output1=mu+sigma*T1*factor;

	return output1;
}


void VehicleGeneration()
//��������
{
	srand(int(time(0)));//�������������
	float randn;//�����С
	randn=rand();

	for (int n=0;n<NodeNumTol;n++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			for (int l=0;l<NumofLanes;l++)
			{
				if (SimulationClock==0)//�趨���濪ʼʱ����ĿΪ��
				{
					srand(unsigned(randn));
					randn=rand();

					float VehArriveRate=RoadNetworkRes[n].VehicleArrRate[r];//������������ͬһ������������

					interval[n][r][l]=RandomGen(VehArriveRate);	
					intervalReal[n][r][l]=int(interval[n][r][l]+1);//ת��Ϊ��������
					VehGenEvent[n][r][l]=SimulationClock+intervalReal[n][r][l];//������һ���������ʱ��

				}
				else
				{
					if (VehGenEvent[n][r][l]==SimulationClock)//�������������¼�
					{
						//****(1) ��������Ҫʵ�����������󣬲������·����
						srand(unsigned(randn));
						randn=rand();
						//���ɳ���
						VehicleAgent AgentCar;
						AgentCar.position=0;//·���ܳ���Ϊ1000�ף���ʼ����λ��Ϊ0��ע��Ҫ�����ڳ�ʼ��λ��-1
						AgentCar.speed=10;//float(abs(GaussRandomGen(40/3.6,0.0001)));//�ٶ�ƽ��ֵΪ40km/Сʱ
						AgentCar.acceleration=0;//�趨��ʼ���ٶ�Ϊ0�������ٽ��룬����Ҳ���Է��ϸ�˹�ֲ�����				
						AgentCar.ExpSpeed=60/3.6+(20/3.6)*((float)rand()/RAND_MAX);

						AgentCar.RoadTimeConsumed=0;
						int temp2;
						temp2=RoadNetworkRes[n].NodeLinkTo[r];

						bool yesornot;
						// wangkai
						if ((n==1&&r==1)||(n==5&&r==0)||(n==3&&r==3)||(n==9&&r==0)||(n==15&&r==2)||(n==21&&r==1)||(n==19&&r==2)||(n==23&&r==3))//(n==7&&r==2)||(n==11&&r==3)||(n==13&&r==1)||(n==17&&r==0)/
						{

							int rear=RoadNetworkRes[n].Road[r][l].RoadRear;
							int front=RoadNetworkRes[n].Road[r][l].RoadFront;
							if ((rear+1)%RoadSIZE!=front)
							{
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].position=AgentCar.position;
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].speed=AgentCar.speed;
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].acceleration=AgentCar.acceleration;
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].ExpSpeed=AgentCar.ExpSpeed;
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].RoadTimeConsumed=AgentCar.RoadTimeConsumed;
								RoadNetworkRes[n].Road[r][l].BufferRoad[rear].VehicleFlag=true;
								RoadNetworkRes[n].Road[r][l].RoadRear=(RoadNetworkRes[n].Road[r][l].RoadRear+1)%RoadSIZE;//ѭ�����еĲ���
								RoadNetworkRes[n].Road[r][l].RoadCount++;
							}

						}
						//****(2) ������һ�γ��������ʱ��
						float VehArriveRate=RoadNetworkRes[n].VehicleArrRate[r];//������������ͬһ������������
						interval[n][r][l]=RandomGen(VehArriveRate);
						intervalReal[n][r][l]=int(interval[n][r][l]+1);//ת��Ϊ��������
						VehGenEvent[n][r][l]=SimulationClock+intervalReal[n][r][l];
					}
					else
					{
						VehGenEvent[n][r][l]=VehGenEvent[n][r][l];
					}
				}

			}
		}
	}
}



void SimulationProcess()
//�������庯��
{
	float positionNew,speedNew,accelerationNew;
	bool flagNew;

	for (int n=0;n<NodeNumTol;n++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			for (int l=0;l<NumofLanes;l++)
			{
				for (int v=0;v<RoadSIZE;v++)
				{
				
					positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
					speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed;
					accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
					flagNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].VehicleFlag;

					if (RoadNetworkRes[n].Road[r][l].BufferRoad[v].VehicleFlag==true)
					{
						if (RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed==0)
						{
							int a=0;
						}


						//����ͷ��״̬����1
						if(v>0 && RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].VehicleFlag==true)
						{
							if (RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position) //��������
							{
								positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
								speedNew=0;
								accelerationNew=0;
							}
							if (RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position<=150 && RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>0)
								//����״̬
							{
								if (RoadNetworkRes[n].Road[r][l].BufferRoad[v].position==0)//�շ����ĳ���
								{
									if (RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position>0) //ǰ���Ѵ�������״̬�����η�����Ҫ�����ʼ�ٶȹ���
									{
										while(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1>RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
											//���������ٶ�
										{
											RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed/2;
										}
										//���¼��ٶ�
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>5)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].speed-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);

										//�����ٶ�
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
										if (speedNew<0)
										{
											speedNew=0;
											accelerationNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
										if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
											speedNew=0;
											accelerationNew=0;
										}
									}
									else//ǰ����δ�������������ֳ�ʼ״̬����
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										speedNew=0;
										accelerationNew=0;
									}
								}
								else//�Ǹշ�������
								{
									//���¼��ٶ�
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>5)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].speed-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);

									//�����ٶ�
									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
									if (speedNew<0)
									{
										speedNew=0;
										accelerationNew=0;
									}

									//����λ��
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
									if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										speedNew=0;
										accelerationNew=0;
									}
								}
							}
							if(RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)
								//������ʻ
							{
								//���¼��ٶ�
								if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
									accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
								else
								{
									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
									accelerationNew=0;
								}


								//�����ٶ�
								speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
								if (speedNew<0)
								{
									speedNew=0;
									accelerationNew=0;
								}

								//����λ��
								positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
								if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].acceleration)
								{
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									speedNew=0;
									accelerationNew=0;
								}
							}

						}





						//������ͷ��״̬���·���2
						if (v==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].VehicleFlag==true)
						{
							if (RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position)
								//��������
							{
								positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
								speedNew=0;
								accelerationNew=0;
							}
							if (RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position<=150 && RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>0)
								//����״̬
							{
								if (RoadNetworkRes[n].Road[r][l].BufferRoad[v].position==0)//�շ����ĳ���
								{
									if (RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position>0)//ǰ���Ѵ�������״̬�����η�����Ҫ�����ʼ�ٶȹ���
									{
										while(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1>RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position)
											//���������ٶ�
										{
											RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed/2;
										}
										//���¼��ٶ�
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>5)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].speed-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);

										//�����ٶ�
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
										if (speedNew<0)
										{
											speedNew=0;
											accelerationNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
										if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
											speedNew=0;
											accelerationNew=0;
										}
									}
									else//ǰ����δ�������������ֳ�ʼ״̬����
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										speedNew=0;
										accelerationNew=0;
									}
								}
								else//�Ǹշ�������
								{
									//���¼��ٶ�
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>5)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].speed-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);

									//�����ٶ�
									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
									if (speedNew<0)
									{
										speedNew=0;
										accelerationNew=0;
									}

									//����λ��
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
									if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										speedNew=0;
										accelerationNew=0;
									}
								}
							}
							if(RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)
								//������ʻ
							{
								//���¼��ٶ�
								if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
									accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
								else
								{
									speedNew=speedNew;
									accelerationNew=0;
								}


								//�����ٶ�
								speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
								if (speedNew<0)
								{
									speedNew=0;
									accelerationNew=0;
								}

								//����λ��
								positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;
								if (positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position || positionNew>=RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].position+RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration*RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].acceleration)
								{
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									speedNew=0;
									accelerationNew=0;
								}
							}
						}




						//����ͷ��״̬���·���
						if ((v>0 && RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].VehicleFlag==false)  || (v==0 && RoadNetworkRes[n].Road[r][l].BufferRoad[RoadSIZE-1].VehicleFlag==false))
						{
							if (RoadNetworkRes[n].TrafficLight.GreenLightStatus_Phase1==true)
							{
								if ((r%2==0 && l==1) || l==2)
								{
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else

									{
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
										accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ
									}


									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
									if(speedNew<0)
									{
										speedNew=0;
									}
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
									if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									}
								}
								else//�������Ϊ���
								{
									if(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)//�����·�ܳ������Ϊ0�����·�ܳ�Ϊ��ͨ������λ��
									{
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else

										{
											speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
											accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ
										}


										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}
									}
									else
									{
										//����GMģ�͸��¼��ٶ�
										accelerationNew=1*RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(0-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);//1Ϊ��ӳǿ��ϵ��					

										//�����ٶ�				
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}	
									}
								}
							}
							if (RoadNetworkRes[n].TrafficLight.GreenLightStatus_Phase2==true)
							{
								if ((r%2==0 && l==0) || l==2)
								{
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else
									{
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
										accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ
									}


									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
									if(speedNew<0)
									{
										speedNew=0;
									}
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
									if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									}
								}
								else
								{
									if(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)//�����·�ܳ������Ϊ0�����·�ܳ�Ϊ��ͨ������λ��
									{
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else

										{
											speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
											accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ
										}


										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}
									}
									else
									{
										//����GMģ�͸��¼��ٶ�
										accelerationNew=1*RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(0-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);//1Ϊ��ӳǿ��ϵ��					

										//�����ٶ�				
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}	

									}

								}
							}
							if (RoadNetworkRes[n].TrafficLight.GreenLightStatus_Phase3==true)
							{
								if ((r%2!=0 && l==1) || l==2)
								{
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else
									{
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
										accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ
									}


									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
									if(speedNew<0)
									{
										speedNew=0;
									}
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
									if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									}
								}
								else
								{
									if(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)//�����·�ܳ������Ϊ0�����·�ܳ�Ϊ��ͨ������λ��
									{
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else

										{
											speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed;
											accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ

										}

										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}
									}
									else
									{
										//����GMģ�͸��¼��ٶ�
										accelerationNew=1*RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(0-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);//1Ϊ��ӳǿ��ϵ��					

										//�����ٶ�				
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}	

									}

								}
							}
							if (RoadNetworkRes[n].TrafficLight.GreenLightStatus_Phase4==true)
							{
								if ((r%2!=0 && l==0) || l==2)
								{
									if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
										accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
									else
										accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ

									speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
									if(speedNew<0)
									{
										speedNew=0;
									}
									positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
									if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
									{
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
									}
								}
								else
								{
									if(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position>150)//�����·�ܳ������Ϊ0�����·�ܳ�Ϊ��ͨ������λ��
									{
										if(RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed<RoadNetworkRes[n].Road[r][l].BufferRoad[v].ExpSpeed)
											accelerationNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration+1;
										else
											accelerationNew=0;//�ٶȴﵽ�����ٶȺ�ת��������ʻ

										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*1+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��	
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v-1].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}
									}
									else
									{
										//����GMģ�͸��¼��ٶ�
										accelerationNew=1*RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed*(0-RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed)/(VehicleLength*RoadSIZE-RoadNetworkRes[n].Road[r][l].BufferRoad[v].position);//1Ϊ��ӳǿ��ϵ��					

										//�����ٶ�				
										speedNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration*1;
										if(speedNew<0)
										{
											speedNew=0;
										}

										//����λ��
										positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position+RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed+0.5*RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration;//ʱ����Ϊ1��
										if(positionNew<RoadNetworkRes[n].Road[r][l].BufferRoad[v].position)
										{
											positionNew=RoadNetworkRes[n].Road[r][l].BufferRoad[v].position;
										}	

									}

								}
							}
						}

						//�����ݸ�ֵ

						RoadNetworkRes[n].Road[r][l].BufferRoad[v].position=positionNew;
						RoadNetworkRes[n].Road[r][l].BufferRoad[v].speed=speedNew;
						RoadNetworkRes[n].Road[r][l].BufferRoad[v].acceleration=accelerationNew;		
					
						if (positionNew>0)
						{
							RoadNetworkRes[n].Road[r][l].BufferRoad[v].RoadTimeConsumed=RoadNetworkRes[n].Road[r][l].BufferRoad[v].RoadTimeConsumed+1;
						}
					}
				}
			}
		}
	}
}


int GetODList(int Roadnum, int Lanenum)
{
	int destinationRoadNum;
	int i,j;
	i=Roadnum;
	j=Lanenum;
	if (i==0)//��
	{
		if (j==0)//��ת
		{
			destinationRoadNum=3;
		}
		if (j==1)//ֱ��
		{
			destinationRoadNum=2;
		}
		if (j==2)//��ת
		{
			destinationRoadNum=1;
		}
	}
	if (i==1)//��
	{
		if (j==0)
		{
			destinationRoadNum=0;
		}
		if (j==1)
		{
			destinationRoadNum=3;
		}
		if (j==2)
		{
			destinationRoadNum=2;
		}
	}
	if (i==2)//��
	{
		if (j==0)
		{
			destinationRoadNum=1;
		}
		if (j==1)
		{
			destinationRoadNum=0;
		}
		if (j==2)
		{
			destinationRoadNum=3;
		}
	}
	if (i==3)//��
	{
		if (j==0)
		{
			destinationRoadNum=2;
		}
		if (j==1)
		{
			destinationRoadNum=1;
		}
		if (j==2)
		{
			destinationRoadNum=0;
		}
	}
	return destinationRoadNum;
}

void DataPorcess()
//ÿ������ʱ�̵ķ������ݺ���
{
	int destinationRoad=-1;
	int NodeNumTemp=-1;
	int RoadNumTemp=-1;
	int LaneNumTemp=-1;
	VehicleAgent VehicleTemp;

	srand(int(time(0)));

	for (int n=0;n<NodeNumTol;n++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			for (int l=0;l<NumofLanes;l++)
			{

				destinationRoad=GetODList(r,l);//���ڵ��ڵ�Ŀ��·��
				if (RoadNetworkRes[n].NodeLinkTo[destinationRoad]==-1)
				//ʻ��·�� ɾ������
				{
					int front=RoadNetworkRes[n].Road[r][l].RoadFront;
					if (RoadNetworkRes[n].Road[r][l].BufferRoad[front].position>RoadSIZE*VehicleLength+1)
					{
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].position=-1;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].speed=0;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].acceleration=0;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].VehicleFlag=false;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].ExpSpeed=0;

						RoadNetworkRes[n].Road[r][l].RoadFront=(RoadNetworkRes[n].Road[r][l].RoadFront+1)%RoadSIZE;
						RoadNetworkRes[n].Road[r][l].RoadCount--;

						RoadNetworkRes[n].RoadTraveTime[r]=RoadNetworkRes[n].RoadTraveTime[r]+RoadNetworkRes[n].Road[r][l].BufferRoad[front].RoadTimeConsumed;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].RoadTimeConsumed=0;

						RoadNetworkRes[n].VehicleInNumInAll[r]=RoadNetworkRes[n].VehicleInNumInAll[r]+1;

						RoadNetworkRes[n].RoadVehiclePassNum[r][l]=RoadNetworkRes[n].RoadVehiclePassNum[r][l]+1;
					}		
					
				}
				else
				//�ӱ���·ʻ��������·
				{
					int front=RoadNetworkRes[n].Road[r][l].RoadFront;
					if (RoadNetworkRes[n].Road[r][l].BufferRoad[front].position>RoadSIZE*VehicleLength+1)
					{
						NodeNumTemp=RoadNetworkRes[n].NodeLinkTo[destinationRoad];//Ŀ��ڵ��
						RoadNumTemp=RoadNetworkRes[n].RoadLinkTo[destinationRoad];//Ŀ��ڵ��Ŀ���·
						
						float temp=(float)rand()/RAND_MAX;
						if (temp<=0.5 && temp>=0)
						{
							LaneNumTemp=1;
						}
						if (temp>0.5 && temp<=0.8)
						{
							LaneNumTemp=0;
						}
						if (temp>0.8)
						{
							LaneNumTemp=2;
						}
						

						int front=RoadNetworkRes[n].Road[r][l].RoadFront;
						VehicleTemp=RoadNetworkRes[n].Road[r][l].BufferRoad[front];

						//ԭ��������ɾ��
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].position=-1;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].speed=0;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].acceleration=0;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].VehicleFlag=false;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].ExpSpeed=0;

						RoadNetworkRes[n].Road[r][l].RoadFront=(RoadNetworkRes[n].Road[r][l].RoadFront+1)%RoadSIZE;
						RoadNetworkRes[n].Road[r][l].RoadCount--;

						RoadNetworkRes[n].RoadTraveTime[r]=RoadNetworkRes[n].RoadTraveTime[r]+RoadNetworkRes[n].Road[r][l].BufferRoad[front].RoadTimeConsumed;
						RoadNetworkRes[n].Road[r][l].BufferRoad[front].RoadTimeConsumed=0;

						RoadNetworkRes[n].VehicleInNumInAll[r]=RoadNetworkRes[n].VehicleInNumInAll[r]+1;

						RoadNetworkRes[n].RoadVehiclePassNum[r][l]=RoadNetworkRes[n].RoadVehiclePassNum[r][l]+1;

						//�����µ�Ŀ���·(rear+1)%RoadSIZE!=front
						if ((RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadRear+1)%RoadSIZE!=RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadFront)
						{
							VehicleTemp.position=0;
							VehicleTemp.speed=VehicleTemp.speed/2;
							VehicleTemp.acceleration=0;
							VehicleTemp.RoadTimeConsumed=0;


							int rear=RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadRear;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].position=VehicleTemp.position;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].speed=VehicleTemp.speed;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].acceleration=VehicleTemp.acceleration;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].RoadTimeConsumed=VehicleTemp.RoadTimeConsumed;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].ExpSpeed=VehicleTemp.ExpSpeed;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].BufferRoad[rear].VehicleFlag=true;
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadRear=(RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadRear+1)%RoadSIZE;//ѭ�����еĲ���
							RoadNetworkRes[NodeNumTemp].Road[RoadNumTemp][LaneNumTemp].RoadCount++;
						}
						
					}
					
				}

			}
		}
	}
	
}

void DateProcessEnd()
//���յ����ݴ������������ݣ�����·�ι�����
{
	for (int n=0;n<NodeNumTol;n++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			/*//����t
			RoadNetworkRes[n].RoadTraveTime[r]=(float)(RoadNetworkRes[n].RoadTraveTime[r]/(60*RoadNetworkRes[n].VehicleInNumInAll[r]));
			//����Qmax
			int temp=-1;
			for (int l=0;l<NumofLanes;l++)
			{
				if (RoadNetworkRes[n].RoadVehiclePassNum[r][l]>temp)
				{
					RoadNetworkRes[n].VehicleInNumInMax[r]=RoadNetworkRes[n].RoadVehiclePassNum[r][l];
					temp=RoadNetworkRes[n].RoadVehiclePassNum[r][l];
				}
			}
			//����Qtotal
			RoadNetworkRes[n].VehicleInNumInAll[r]=(int)(RoadNetworkRes[n].VehicleInNumInAll[r]/NumofLanes);
			float ratio=(float)RoadNetworkRes[n].VehicleInNumInMax[r]/RoadNetworkRes[n].VehicleInNumInAll[r];

			/////////�����·�ι�����
			RoadNetworkRes[n].RoadRelationNum[r]=(0.5/(1+RoadNetworkRes[n].RoadTraveTime[r]))*(ratio-1);
			//RoadNetworkRes[n].RoadRelationNum[r]=(ratio-(1/(1+RoadNetworkRes[n].RoadTraveTime[r])))/2;
			if (RoadNetworkRes[n].RoadRelationNum[r]>1)
			{
				RoadNetworkRes[n].RoadRelationNum[r]=1;
			}*/

			RoadNetworkRes[n].RoadRelationNum[r]=(float)RoadNetworkRes[n].VehicleInNumInAll[r]/(2*RoadSIZE*VehicleLength*NumofLanes);
			if (RoadNetworkRes[n].RoadRelationNum[r]>1)
			{
				RoadNetworkRes[n].RoadRelationNum[r]=1;
			}

		}
	}

	//��дlaplace����
	for (int n=0;n<NodeNumTol;n++)
	{
		for (int r=0;r<NumofRoads;r++)
		{
			int Dn=RoadNetworkRes[n].NodeLinkTo[r];
			
			if (Dn!=-1)
			{
				LapMatirx[n][Dn]=RoadNetworkRes[n].RoadRelationNum[r];
			}
		}
	}

	for (int i=0;i<NodeNumTol;i++)
	{
		for (int j=0;j<NodeNumTol;j++)
		{
			if (LapMatirx[i][j]>LapMatirx[j][i])
			{
				LapMatirx[j][i]=LapMatirx[i][j];
			}
			else
			{
				LapMatirx[i][j]=LapMatirx[j][i];
			}
		}
	}

	

	for (int n=0;n<NodeNumTol;n++)
	{
		float count=0;
		float temp=0;
		for(int k=0;k<NodeNumTol;k++)
		{
			if (LapMatirx[n][k]!=-1)
			{
				count=count+1;
				temp=temp+LapMatirx[n][k];
			}
		}
		Paint[n]=(float)temp/count;
	}


	//д��ͼ����
	FILE * PaintData;
	PaintData=fopen("PaintData.txt","w+");

	for (int i=0;i<NodeNumTol;i++)
	{	
		if (i>0 && i%RowSize==0)
		{
			fprintf(PaintData,"\n");

		}
		fprintf(PaintData,"%f ",Paint[i]);
	}
	
	fclose(PaintData);

	//д�����������ļ�
	FILE * RelationData;
	RelationData=fopen("RelationData.txt","w+");

	for (int n=0;n<NodeNumTol;n++)
	{	
		for (int k=0;k<NodeNumTol;k++)
		{
			if (LapMatirx[n][k]!=-1 && k>n)
			{	
				fprintf(RelationData,"%d--%d : %f \n",n,k,LapMatirx[n][k]);
			}	
		}
		
	}

	fclose(RelationData);


}


void PeopleInitial()
{
	
	srand(time(0));
	for (int i=0;i<PeopleNum;i++)
	{
		//����
		PeopleCurrent[i].PeopleIndex=i;
		for (int j=0;j<RoadNum;j++)
		{
			PeopleCurrent[i].Chomosome[j]=rand()%2;
		}
		for (int ii=0;ii<NodeNumTol;ii++)
		{
			for (int jj=0;jj<NodeNumTol;jj++)
			{
				if (ii>=jj)
				{
					PeopleCurrent[i].LapMatrixBinary[ii][jj]=-1;
					PeopleNew[i].LapMatrixBinary[ii][jj]=-1;
				}
				else
				{
					PeopleCurrent[i].LapMatrixBinary[ii][jj]=0;
					PeopleNew[i].LapMatrixBinary[ii][jj]=0;
				}
			}
		}

		//���ַ�����Ӧ��������Ŀ����ֵ
		PeopleCurrent[i].N=0;
		PeopleNew[i].N=0;
		//���ַ�����Ӧ��������Ӧ�����鸶��ֵ
		for(int sa=0;sa<NodeNumTol;sa++)	
		{
			for (int sa2=0;sa2<NodeNumTol;sa2++)
			{
				PeopleCurrent[i].NRelation[sa][sa2]=-1;
				PeopleNew[i].NRelation[sa][sa2]=-1;
			}			
		}
		
		//���ַ�����Ӧ�ȸ���ֵ
		PeopleCurrent[i].fitness=0;
		PeopleNew[i].fitness=0;


		//����Ⱦɫ�������
		for (int j=0;j<RoadNum;j++)
		{
			if (LapVector[j]<0.2)
			{
				PeopleCurrent[i].Chomosome[j]=0;
			}
			if (LapVector[j]>0.8)
			{
				PeopleCurrent[i].Chomosome[j]=1;
			}
		}


	}

}


void EvaluatePeopleCurrent()
{
	for (int i=0;i<NodeNumTol;i++)
	{
		for (int j=0;j<NodeNumTol;j++)
		{
			LapMatirx2[i][j]=LapMatirx[i][j];
		}
	}

	//�����뻮�ַ���������صı�����ʼ��
	for (int i=0;i<PeopleNum;i++)
	{
		//���ַ�����Ӧ��������Ŀ����ֵ
		PeopleCurrent[i].N=0;
		//���ַ�����Ӧ��������Ӧ�����鸶��ֵ
		for(int sa=0;sa<NodeNumTol;sa++)	
		{
			for (int sa2=0;sa2<NodeNumTol;sa2++)
			{
				PeopleCurrent[i].NRelation[sa][sa2]=-1;
			}

		}
		//���ַ�����Ӧ�ȸ���ֵ
		PeopleCurrent[i].fitness=0;
	}


	int TempIdx=0;
	//����
	for (int i=0;i<PeopleNum;i++)
	{
		TempIdx=0;
		for (int ii=0;ii<NodeNumTol;ii++)
		{
			for (int jj=0;jj<NodeNumTol;jj++)
			{
				if (jj>ii && LapMatirx2[ii][jj]!=-1)
				{
					PeopleCurrent[i].LapMatrixBinary[ii][jj]=PeopleCurrent[i].Chomosome[TempIdx];
					TempIdx++;
				}
			}
		}
	}

	int a=0;

	//��PeopleCurrent[i].LapMatrixBinary[ii][jj]Ϊ�����б����������Լ���������� �Ӷ��ó�����������ֵ
	bool CheckFlag=true;
	int roottemp=-1;
	int treeNumber;
	int root;

	int TreeIdx;
	//�洢���ڵ��Ҷ�ڵ����ݵ�һά������������
	int LeafIdx;
	//ȷ����ǰ���ĸ��ڵ�			
	int rootIdx;

	for (int i=0;i<PeopleNum;i++)
	{
		for (int nf=0;nf<NodeNumTol;nf++)
		{
			NodeFlag[nf]=true;
			LeafFlag[nf]=true;
		}
		CheckFlag=true;


		treeNumber=0;
		root=100000;
		
		while (root!=-1)//��������������ͼ���
		{
			//���������Ļ�����Ŀ
			//PeopleCurrent[i].N=PeopleCurrent[i].N+1;

			//һ��ѭ������һ���� �洢�ṹΪһά����tree

			for (int mm=0;mm<NodeNumTol;mm++)
			{
				tree[mm].NodeIndex=-1;
				for (int mml=0;mml<4;mml++)
				{
					tree[mm].Leaf[mml]=-1;
				}
				
			}
			
			for(int Tidx=0;Tidx<NodeNumTol;Tidx++)
			{
				for (int Lidx=0;Lidx<4;Lidx++)
				{
					tree[Tidx].NodeIndex=-1;
					tree[Tidx].Leaf[Lidx]=-1;
				}
			}


			//�洢�����нڵ����ݵ�һά������������
			 TreeIdx=1;
			//�洢���ڵ��Ҷ�ڵ����ݵ�һά������������
			 LeafIdx=0;
			//ȷ����ǰ���ĸ��ڵ�			
			rootIdx=0;
			root=-1;
			for (int k=0;k<NodeNumTol;k++)
			{
				if (NodeFlag[k]==true)
				{
					root=k;
					break;
				}
			}
			tree[rootIdx].NodeIndex=root;
			roottemp=root;
			
			// Ѱ�ҵ����������нڵ�
			while (rootIdx<TreeIdx)
			{
				// ���ڵ��Ӧ��Ҷ�ڵ���Ÿ�ֵ
				for (int k=0;k<NodeNumTol;k++)
				{
					if (PeopleCurrent[i].LapMatrixBinary[roottemp][k]==1 && NodeFlag[k]==true)
					{	
						tree[rootIdx].Leaf[LeafIdx]=k;
						LeafIdx++;
					}
				}
				//�ҵ���ǰroot���е�leaf֮��root��־״̬����
				NodeFlag[roottemp]=false;
				
				//��root��leaf�ڵ����tree����
				int leaftemp=-1;
				for (int k=0;k<4;k++)
				{
					if (tree[rootIdx].Leaf[k]!=-1 && LeafFlag[tree[rootIdx].Leaf[k]]==true)
					{
						leaftemp=tree[rootIdx].Leaf[k];
						
						tree[TreeIdx].NodeIndex=leaftemp;
						TreeIdx++;

						LeafFlag[leaftemp]=false;
					}
				}

				//�ض�λroot
				rootIdx++;
				if (rootIdx==TreeIdx)
				{
					NodeFlag[tree[TreeIdx].NodeIndex]=false;
				}

				if (tree[rootIdx].NodeIndex!=-1)
				{
					roottemp=tree[rootIdx].NodeIndex;
				}				
			}
			

			
			if (root!=-1)
			{
				for (int leafCount=0;leafCount<NodeNumTol;leafCount++)
				{
					if (tree[leafCount].NodeIndex!=-1)
					{
						PeopleCurrent[i].NRelation[treeNumber][leafCount]=tree[leafCount].NodeIndex;
					}
				}

				treeNumber=treeNumber+1;
			}
		}


		//���㲻ͬ���ַ�������
		PeopleCurrent[i].N=treeNumber;


		int AreaNumber=treeNumber;
		float AreaScore=0;
		
		for (int tempindex=0;tempindex<NodeNumTol;tempindex++)
		{
			temp[tempindex]=temp2[tempindex]=temp3[tempindex]=-1;
		}


		int temp2index=0;
		for (int nn=0;nn<AreaNumber;nn++)
		{
			
			for (int n2=0;n2<NodeNumTol;n2++)
			{
				temp[n2]=PeopleCurrent[i].NRelation[nn][n2];
			}

			//��temp���и�ֵ������
			temp2index=0;
			for (int n3=0;n3<NodeNumTol;n3++)
			{
				int x=temp[n3];
				int y=-1;
				for (int n4=n3+1;n4<NodeNumTol;n4++)
				{
					y=temp[n4];
					if (x!=-1 && y!=-1)
					{
						if (LapMatirx2[x][y]!=-1)
						{
							temp2[temp2index]=LapMatirx2[x][y];
							temp2index++;
						}
						
					}
					
				}	
			}
			//����
			for (int a=0;a<NodeNumTol;a++)
			{
				float mark=10000;
				int markIdx=-1;
				for (int a1=0;a1<NodeNumTol;a1++)
				{
					if (temp2[a1]<mark  &&  temp2[a1]>0)
					{
						mark=temp2[a1];
						markIdx=a1;
					}
				}
				temp3[a]=mark;
				temp2[markIdx]=-1;
			}

			//�������������Ϲ�����
			float SubareaScore=1;
			for (int a=0;a<NodeNumTol;a++)
			{
				
				if (temp3[a]!=-1 && temp3[a]!=10000)
				{
					if (a==0)
					{
						SubareaScore=SubareaScore*pow(temp3[a],1);
					}
					else
					{
						SubareaScore=SubareaScore*pow(temp3[a],1/a);
					}
					
				}
			}
			
			if (SubareaScore==1 || SubareaScore==10000)
			{
				SubareaScore=0;
			}

			AreaScore=AreaScore+SubareaScore;
			
		}

		PeopleCurrent[i].fitness=-pow(PeopleCurrent[i].N,2.0)+AreaScore;
	}

	int aa=0;

}

void CrossoverAndMutationOperation()
{
	srand(time(0));
	//����
	for (int i=0;i<PeopleNum;i++)
	{
		float pc=(float)rand()/RAND_MAX;
		if (pc<PCross)
		{
			int anotherPlan=rand()%PeopleNum;
			int crosspoint=rand()%RoadNum;
			
			for (int j=0;j<crosspoint;j++)
			{
				PeopleNew[i].Chomosome[j]=PeopleCurrent[i].Chomosome[j];
			}
			for (int j=crosspoint;j<RoadNum;j++)
			{
				PeopleNew[i].Chomosome[j]=PeopleCurrent[anotherPlan].Chomosome[j];
			}
			
		}
	}
	//����
	for (int i=0;i<PeopleNum;i++)
	{
		for (int j=0;j<RoadNum;j++)
		{
			float pm=(float)rand()/RAND_MAX;
			if (pm<PMutate)
			{
				if (PeopleNew[i].Chomosome[j]==0)
				{
					PeopleNew[i].Chomosome[j]=1;
				}
				else
				{
					PeopleNew[i].Chomosome[j]=0;
				}
			}
			
		}
		
	}

	for (int i=0;i<PeopleNum;i++)
	{
		//����Ⱦɫ�������
		for (int j=0;j<RoadNum;j++)
		{
			if (LapVector[j]<0.2)
			{
				PeopleNew[i].Chomosome[j]=0;
			}
			if (LapVector[j]>0.8)
			{
				PeopleNew[i].Chomosome[j]=1;
			}
		}

	}

	

}

void EvaluatePeopleNew()
{
	/*for (int i=0;i<NodeNumTol;i++)
	{
		for (int j=0;j<NodeNumTol;j++)
		{
			LapMatirx2[i][j]=LapMatirx[i][j];
		}
	}*/

	//�����뻮�ַ���������صı�����ʼ��
	for (int i=0;i<PeopleNum;i++)
	{
		//���ַ�����Ӧ��������Ŀ����ֵ
		PeopleNew[i].N=0;
		//���ַ�����Ӧ��������Ӧ�����鸳��ֵ
		for(int sa=0;sa<NodeNumTol;sa++)	
		{
			for (int sa2=0;sa2<NodeNumTol;sa2++)
			{
				PeopleNew[i].NRelation[sa][sa2]=-1;
			}
		}
		//���ַ�����Ӧ�ȸ���ֵ
		PeopleNew[i].fitness=0;
	}


	int TempIdx=0;
	//����
	for (int i=0;i<PeopleNum;i++)
	{
		TempIdx=0;
		for (int ii=0;ii<NodeNumTol;ii++)
		{
			for (int jj=0;jj<NodeNumTol;jj++)
			{
				if (jj>ii && LapMatirx2[ii][jj]!=-1)
				{
					PeopleNew[i].LapMatrixBinary[ii][jj]=PeopleNew[i].Chomosome[TempIdx];
					TempIdx++;
				}
			}
		}
	}

	int a=0;

	//��PeopleCurrent[i].LapMatrixBinary[ii][jj]Ϊ�����б����������Լ���������� �Ӷ��ó�����������ֵ
	for (int NodeFlagIdx=0;NodeFlagIdx<NodeNumTol;NodeFlagIdx++)
	{
		NodeFlag[NodeFlagIdx]=true;
	}

	bool CheckFlag=true;
	int roottemp=-1;
	int root;
	int treeNumber;

	//�洢�����нڵ����ݵ�һά������������
	int TreeIdx;
	//�洢���ڵ��Ҷ�ڵ����ݵ�һά������������
	int LeafIdx;
	//ȷ����ǰ���ĸ��ڵ�			
	int rootIdx;

	for (int i=0;i<PeopleNum;i++)
	{
		for (int nf=0;nf<NodeNumTol;nf++)
		{
			NodeFlag[nf]=true;
			LeafFlag[nf]=true;
		}
		CheckFlag=true;


		treeNumber=0;
		root=100000;
		
		while (root!=-1)//��������������ͼ���
		{
			//���������Ļ�����Ŀ
			//PeopleCurrent[i].N=PeopleCurrent[i].N+1;

			//һ��ѭ������һ���� �洢�ṹΪһά����tree
			for (int mm=0;mm<NodeNumTol;mm++)
			{
				tree[mm].NodeIndex=-1;
				for (int mml=0;mml<4;mml++)
				{
					tree[mm].Leaf[mml]=-1;
				}
			}

			for(int Tidx=0;Tidx<NodeNumTol;Tidx++)
			{
				for (int Lidx=0;Lidx<4;Lidx++)
				{
					tree[Tidx].NodeIndex=-1;
					tree[Tidx].Leaf[Lidx]=-1;
				}
			}


			//�洢�����нڵ����ݵ�һά������������
			TreeIdx=1;
			//�洢���ڵ��Ҷ�ڵ����ݵ�һά������������
			LeafIdx=0;
			//ȷ����ǰ���ĸ��ڵ�			
			rootIdx=0;
			root=-1;
			for (int k=0;k<NodeNumTol;k++)
			{
				if (NodeFlag[k]==true)
				{
					root=k;
					break;
				}
			}
			tree[rootIdx].NodeIndex=root;
			roottemp=root;

			// Ѱ�ҵ����������нڵ�
			while (rootIdx<TreeIdx)
			{
				// ���ڵ��Ӧ��Ҷ�ڵ���Ÿ�ֵ
				for (int k=0;k<NodeNumTol;k++)
				{
					if (PeopleNew[i].LapMatrixBinary[roottemp][k]==1 && NodeFlag[k]==true)
					{	
						tree[rootIdx].Leaf[LeafIdx]=k;
						LeafIdx++;
					}
				}
				//�ҵ���ǰroot���е�leaf֮��root��־״̬����
				NodeFlag[roottemp]=false;

				//��root��leaf�ڵ����tree����
				int leaftemp=-1;
				for (int k=0;k<4;k++)
				{
					if (tree[rootIdx].Leaf[k]!=-1 && LeafFlag[tree[rootIdx].Leaf[k]]==true)
					{
						leaftemp=tree[rootIdx].Leaf[k];

						tree[TreeIdx].NodeIndex=leaftemp;
						TreeIdx++;

						LeafFlag[leaftemp]=false;
					}
				}

				//�ض�λroot
				rootIdx++;
				if (rootIdx==TreeIdx)
				{
					NodeFlag[tree[TreeIdx].NodeIndex]=false;
				}

				if (tree[rootIdx].NodeIndex!=-1)
				{
					roottemp=tree[rootIdx].NodeIndex;
				}
			}



			if (root!=-1)
			{
				for (int leafCount=0;leafCount<NodeNumTol;leafCount++)
				{
					if (tree[leafCount].NodeIndex!=-1)
					{
						PeopleNew[i].NRelation[treeNumber][leafCount]=tree[leafCount].NodeIndex;
					}
				}

				treeNumber=treeNumber+1;
			}
		}


		//���㲻ͬ���ַ�������
		PeopleNew[i].N=treeNumber;

		int AreaNumber=treeNumber;
		float AreaScore=0;
		
		for (int tempindex=0;tempindex<NodeNumTol;tempindex++)
		{
			temp[tempindex]=temp2[tempindex]=temp3[tempindex]=-1;
		}


		int temp2index=0;
		for (int nn=0;nn<AreaNumber;nn++)
		{
			for (int n2=0;n2<NodeNumTol;n2++)
			{
				temp[n2]=PeopleNew[i].NRelation[nn][n2];
			}

			//��temp���и�ֵ������
			temp2index=0;
			for (int n3=0;n3<NodeNumTol;n3++)
			{
				int x=temp[n3];
				int y=-1;
				for (int n4=n3+1;n4<NodeNumTol;n4++)
				{
					y=temp[n4];
					if (x!=-1 && y!=-1 )
					{
						if (LapMatirx2[x][y]!=-1)
						{
							temp2[temp2index]=LapMatirx2[x][y];
							temp2index++;
						}
						
					}

				}	
			}
			//����
			for (int a=0;a<NodeNumTol;a++)
			{
				float mark=10000;
				int markIdx=-1;
				for (int a1=0;a1<NodeNumTol;a1++)
				{
					if (temp2[a1]<mark  &&  temp2[a1]>0)
					{
						mark=temp2[a1];
						markIdx=a1;
					}
				}
				temp3[a]=mark;
				temp2[markIdx]=-1;
			}

			//�������������Ϲ�����
			float SubareaScore=1;
			for (int a=0;a<NodeNumTol;a++)
			{

				if (temp3[a]!=-1 && temp3[a]!=10000)
				{
					if (a==0)
					{
						SubareaScore=SubareaScore*pow(temp3[a],1);
					}
					else
					{
						SubareaScore=SubareaScore*pow(temp3[a],1/a);
					}

				}
			}

			if (SubareaScore==1 || SubareaScore==10000)
			{
				SubareaScore=0;
			}

			AreaScore=AreaScore+SubareaScore;

		}


		PeopleNew[i].fitness=-pow(PeopleNew[i].N,2.0)+AreaScore;

	}

}

void SelectionOpration()
{
	for (int i=0;i<PeopleNum;i++)
	{
		if (PeopleNew[i].fitness>PeopleCurrent[i].fitness)
		{
			PeopleCurrent[i]=PeopleNew[i];
		}

	}
}


int main(int argc, char* argv[])
{
///////////////////////////////////////////////////////////
//һ��ͨ���������·���и�·�ι�����
	

	int ComputeMode;
	//0��CPU����ģʽ  ������GPU����ģʽ
	ComputeMode=1;

	//��ʼ��·������
	RoadNetworkInitial();

	//��ͨ�����ݳ�ʼ��
	TrafficLightsDataInitial();
	//·�����ݳ�ʼ��
	RoadNetworkDataInitial();


	//���ɽ�ͨ���Ʒ���
	//TrafficControlProGen();
	TrafficControlProGen2();

	SimulationClock=0;
	SimulationDuration=10800;

	//FILE * recordTime;
	//recordTime=fopen("Count.txt","w+");

	while (SimulationClock<SimulationDuration)
	{
		//1��ˢ�½�ͨ��״̬
		TrafficLightsRefresh();	
		//2�����ճ����ĳ���ʱ�佫�������뵽·���� 
		VehicleGeneration();
		//3��������溯��--���ݡ��������͡�������ԭ�����·���г���״̬
		SimulationProcess();
		//4�����ݺ���--��������λ�õĸı�
		DataPorcess();

		//fprintf(recordTime,"%d \n",RoadNetworkRes[7].Road[2][1].RoadCount);

		SimulationClock++;
		//����Ļ����ʾ��ǰ�ķ���ʱ��
		HANDLE hOut=GetStdHandle(STD_OUTPUT_HANDLE);
		COORD pos0;
		pos0.X=0;
		pos0.Y=0;
		SetConsoleCursorPosition(hOut,pos0);
		fprintf(stdout,"Simulation: CurrentTimestep is %d",SimulationClock);
	}

	
	//fclose(recordTime);

	clock_t start,end;
	float duration;
	start=clock();

	//���յ����ݴ������������ݣ�����·�ι�����
	DateProcessEnd();
	
///////////////////////////////////////////////////////////
//������ʼӦ��GA���п�����������
	int TempIdx=0;
	for (int i=0;i<NodeNumTol;i++)
	{
		for (int j=0;j<NodeNumTol;j++)
		{
			if (LapMatirx[i][j]!=-1  && j>i)//�������Ǿ�����ȡֵ
			{
				LapVector[TempIdx]=LapMatirx[i][j];
				TempIdx++;
			}
		}
	}

	printf("\n");
	printf("Begin to divide the control area into subareas...\n");
	int iteration=0;

	//���ɳ�ʼ��Ⱥ P
	PeopleInitial();
	//������Ⱥ P ��Ӧ�� 
	EvaluatePeopleCurrent();

	FILE * DivideIteration;
	DivideIteration=fopen("DivideIteration.txt","w+");

	while(iteration<MaxGen)
	{
		//����������� W
		CrossoverAndMutationOperation();
		//������Ⱥ W ��Ӧ��
		EvaluatePeopleNew();
		//ѡ����� ���� P �еĻ��ַ�������
		SelectionOpration();

		iteration++;

		HANDLE hOut=GetStdHandle(STD_OUTPUT_HANDLE);
		COORD pos0;
		pos0.X=0;
		pos0.Y=2;
		SetConsoleCursorPosition(hOut,pos0);
		fprintf(stdout,"Division: CurrentIteration is %d",iteration);

		//д�ļ�
		float temp=-1000000;
		int tempmark=-1;
		for(int m=0;m<PeopleNum;m++)
		{
			if (PeopleCurrent[m].fitness>temp)
			{
				temp=PeopleCurrent[m].fitness;
				tempmark=m;
			}
		}
		fprintf(DivideIteration,"%f \n",PeopleCurrent[tempmark].fitness);
	}
	fclose(DivideIteration);

	
	//��ӡ��ѻ��ַ���
	float temp=-1000000;
	int tempmark=-1;
	for(int m=0;m<PeopleNum;m++)
	{
		if (PeopleCurrent[m].fitness>temp)
		{
			temp=PeopleCurrent[m].fitness;
			tempmark=m;
		}
	}

	FILE * Bestdivide;
	Bestdivide=fopen("Bestdivide.txt","w+");
	/*for (int m=0;m<RoadNum;m++)
	{
		fprintf(Bestdivide,"%d ",PeopleCurrent[tempmark].Chomosome[m]);
	}*/

	int TempIdx2=0;
	for (int ii=0;ii<NodeNumTol;ii++)
	{
		//TempIdx2=0;
		for (int jj=0;jj<NodeNumTol;jj++)
		{
			if (jj>ii && LapMatirx[ii][jj]!=-1 )
			{
				PeopleCurrent[tempmark].LapMatrixBinary[ii][jj]=PeopleCurrent[tempmark].Chomosome[TempIdx2];
				TempIdx2++;
			}
		}
	}
	for (int n=0;n<NodeNumTol;n++)
	{	
		for (int k=0;k<NodeNumTol;k++)
		{
			if (PeopleCurrent[tempmark].LapMatrixBinary[n][k]==1 && k>n)
			{	
				fprintf(Bestdivide,"%d--%d : %d \n",n,k,PeopleCurrent[tempmark].LapMatrixBinary[n][k]);
			}
		}
	}
	
	
	fclose(Bestdivide);

	end=clock();
	duration=end-start;

	printf("\nDivide Complete.");

	scanf("hold...");
	return 0;
}
